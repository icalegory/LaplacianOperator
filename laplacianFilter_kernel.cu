#include "hip/hip_runtime.h"
/*
 * This is the CUDA implementation of a close approximation of the morphological
 * Laplacian operator edge detection filter, along with other filters discovered
 * by experimentation.  The CUDA SDK sample Box Filter was used as a base to
 * modify and expand on, and the copyright verbage for the code still present
 * is included below as requested by NVIDIA.
 *
 * ��Ian Calegory, 12/20/2016
 */

////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

#ifndef _LAPLACIANFILTER_KERNEL_H_
#define _LAPLACIANFILTER_KERNEL_H_

//#include <array>
//#include <cstdlib>
#include "laplacianFilter.h"
#include <hip/hip_vector_types.h>
#include <helper_functions.h>

texture<float, 2> tex;
texture<uchar4, 2, hipReadModeNormalizedFloat> rgbaTex;
hipArray *d_array, *d_tempArray;
const int CHANNEL_COUNT = 4;

int disk3x3StructuringElement[] =
{
	0, 1, 0,
	1, 1, 1,
	0, 1, 0
};

int disk5x5StructuringElement[] =
{
	0, 1, 1, 1, 0,
	1, 1, 1, 1, 1,
	1, 1, 1, 1, 1,
	1, 1, 1, 1, 1,
	0, 1, 1, 1, 0
};

int disk7x7StructuringElement[] =
{
	0, 0, 1, 1, 1, 0, 0,
	0, 1, 1, 1, 1, 1, 0,
	1, 1, 1, 1, 1, 1, 1,
	1, 1, 1, 1, 1, 1, 1,
	1, 1, 1, 1, 1, 1, 1,
	0, 1, 1, 1, 1, 1, 0,
	0, 0, 1, 1, 1, 0, 0
};

int square3x3StructuringElement[] =
{
	1, 1, 1,
	1, 1, 1,
	1, 1, 1
};

int square5x5StructuringElement[] =
{
	1, 1, 1, 1, 1,
	1, 1, 1, 1, 1,
	1, 1, 1, 1, 1,
	1, 1, 1, 1, 1,
	1, 1, 1, 1, 1
};

int square7x7StructuringElement[] =
{
	1, 1, 1, 1, 1, 1, 1,
	1, 1, 1, 1, 1, 1, 1,
	1, 1, 1, 1, 1, 1, 1,
	1, 1, 1, 1, 1, 1, 1,
	1, 1, 1, 1, 1, 1, 1,
	1, 1, 1, 1, 1, 1, 1,
	1, 1, 1, 1, 1, 1, 1

};

int ring3x3StructuringElement[] =
{
	0, 1, 0,
	1, 0, 1,
	0, 1, 0
};

int ring5x5StructuringElement[] = 
{
	0, 1, 1, 1, 0,
	1, 0, 0, 0, 1,
	1, 0, 0, 0, 1,
	1, 0, 0, 0, 1,
	0, 1, 1, 1, 0
};

int ring7x7StructuringElement[] =
{
	0, 0, 1, 1, 1, 0, 0,
	0, 1, 0, 0, 0, 1, 0,
	1, 0, 0, 0, 0, 0, 1,
	1, 0, 0, 0, 0, 0, 1,
	1, 0, 0, 0, 0, 0, 1,
	0, 1, 0, 0, 0, 1, 0,
	0, 0, 1, 1, 1, 0, 0
};


// C++11 style arrays are not easy to use in device code
//std::array<std::array<int, 3>, 3> disk3x3StructuringElement{ {
//	{ { 0, 1, 0 } },
//	{ { 1, 1, 1 } },
//	{ { 0, 1, 0 } }
//	} };
// Would be nice to be able to use this or something like it:
//auto &structuringElement = disk5x5StructuringElement;

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line)
{
    if (hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
                file, line, (int)err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// convert floating point rgba color to 32-bit integer
__device__ unsigned int rgbaFloatToInt(float4 rgba)
{
    rgba.x = __saturatef(rgba.x);   // clamp to [0.0, 1.0]
    rgba.y = __saturatef(rgba.y);
    rgba.z = __saturatef(rgba.z);
    rgba.w = __saturatef(rgba.w);
    return ((unsigned int)(rgba.w * 255.0f) << 24) |
           ((unsigned int)(rgba.z * 255.0f) << 16) |
           ((unsigned int)(rgba.y * 255.0f) <<  8) |
           ((unsigned int)(rgba.x * 255.0f));
}

__device__ float4 rgbaIntToFloat(unsigned int c)
{
    float4 rgba;
    rgba.x = (c & 0xff) * 0.003921568627f;       //  /255.0f;
    rgba.y = ((c>>8) & 0xff) * 0.003921568627f;  //  /255.0f;
    rgba.z = ((c>>16) & 0xff) * 0.003921568627f; //  /255.0f;
    rgba.w = ((c>>24) & 0xff) * 0.003921568627f; //  /255.0f;
    return rgba;
}

extern "C"
void initTexture(int width, int height, void *pImage, bool useRGBA)
{
    int size = width * height * (useRGBA ? sizeof(uchar4) : sizeof(float));

    // copy image data to array
    hipChannelFormatDesc channelDesc;
    if (useRGBA)
    {
        channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
    }
    else
    {
        channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    }
    checkCudaErrors(hipMallocArray(&d_array, &channelDesc, width, height));
    checkCudaErrors(hipMemcpyToArray(d_array, 0, 0, pImage, size, hipMemcpyHostToDevice));

    checkCudaErrors(hipMallocArray(&d_tempArray,   &channelDesc, width, height));

    // set texture parameters
    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.filterMode = hipFilterModePoint;
    tex.normalized = true;

    // Bind the array to the texture
    if (useRGBA)
    {
        checkCudaErrors(hipBindTextureToArray(rgbaTex, d_array, channelDesc));
    }
    else
    {
        checkCudaErrors(hipBindTextureToArray(tex, d_array, channelDesc));
    }
}

extern "C"
void freeTextures()
{
    checkCudaErrors(hipFreeArray(d_array));
    checkCudaErrors(hipFreeArray(d_tempArray));
}

// This is used to convert tex2D() call results into the byte components
__device__ void convertTexFloatToUChar(uchar4* dst, const float4 src)
{
	//const unsigned int idx = getTextureIndex();
	//_dst[idx].x = (unsigned char)(_src[idx].x * 255.9999f);
	//_dst[idx].y = (unsigned char)(_src[idx].y * 255.9999f);
	//_dst[idx].z = (unsigned char)(_src[idx].z * 255.9999f);
	//_dst[idx].w = (unsigned char)(_src[idx].w * 255.9999f);

	(*dst).x = (unsigned char)(src.x * 255.9999f);
	(*dst).y = (unsigned char)(src.y * 255.9999f);
	(*dst).z = (unsigned char)(src.z * 255.9999f);
	(*dst).w = (unsigned char)(src.w * 255.9999f);
}

/*
Perform 2D morphological Laplacian operator (approximately? along with a number
of variations) on image using CUDA

This works by calculating the dilation and erosion of the image using the structuring
element centered on the current pixel being processed.  It's passed in as the array
d_structuringElement, which is a 2d array flattened into a 1d array for passing into
CUDA with device hipMemcpyHostToDevice calls.  Dilation is computed by finding the
maximum r, g, and b values for the pixels around the current pixel determined by the
mask of the structuring element.  (If the and of the masking structuring element pixel
and the source image pixel in the corresponding position with the mask overlaid onto
the source image is 1, include that pixel in the source of pixels for choosing maximum
values.)

Erosion is computed similarly, though replacing the source pixel with the components
having the minimum instead of maximum values.

Dilation results in what's called an internal gradient, while erosion results in an
external gradient.  For further reference on computing the internal and external
gradients, see for example http://www.inf.u-szeged.hu/ssip/1996/morpho/morphology.html

The grayscale filter uses the luminosity algorithm for converting to grayscale:
	0.21 R + 0.72 G + 0.07 B

	--Ian Calegory, 12/20/2016

// Comment from original box filter left here for reference--so as a reminder to check
// for coalescence
Note that the x (row) pass suffers from uncoalesced global memory reads,
since each thread is reading from a different row. For this reason it is
better to use texture lookups for the x pass.
The y (column) pass is perfectly coalesced.

Parameters:
id  - pointer to input image in device memory (not used here--texture is used instead)
od  - pointer to destination image in device memory
w   - image width
h   - image height
d_structuringElement - element 0 of the structuring element array
n   - structuring element is nxn matrix

*/
__global__ void
d_laplacianFilter_rgba(unsigned char *id, unsigned char *od, int w, int h, FilterTypeEnum filter, int* d_structuringElement, unsigned int n)
{
	unsigned int colIndex = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int rowIndex = blockIdx.y*blockDim.y + threadIdx.y;

	if (rowIndex < h && colIndex < w) {
		unsigned int index = rowIndex*w*CHANNEL_COUNT + colIndex*CHANNEL_COUNT;
		//if (colIndex > 1085)
		//printf("colIndex, %d, rowIndex, %d, pixelContents, %d, textureContents, %d\n", colIndex, rowIndex, *(id + index), tex2D(rgbaTex, colIndex, rowIndex));
		//printf("w, %d, h, %d, colIndex, %d, rowIndex, %d\n", w, h, colIndex, rowIndex);

		// Convert float4 texture info to uchar4 to extract r, g, b, and a components
		float4 texelCenter = tex2D(rgbaTex, colIndex, rowIndex);
		uchar4 bytesCenterPixel;
		bytesCenterPixel.x = (unsigned char)(texelCenter.x * 255.9999f);
		bytesCenterPixel.y = (unsigned char)(texelCenter.y * 255.9999f);
		bytesCenterPixel.z = (unsigned char)(texelCenter.z * 255.9999f);
		bytesCenterPixel.w = (unsigned char)(texelCenter.w * 255.9999f);
		//printf("r=%d, g=%d, b=%d, a=%d, texel=%d ", bytes.x, bytes.y, bytes.z, bytes.w, texel);
		
		// Now cycle through every pixel of the structuring element, and process
		// both dilation and erosion of the original image.
		unsigned char rMin = 255, gMin = 255, bMin = 255;
		unsigned char rMax = 0, gMax = 0, bMax = 0;
		int maxVert = n / 2;
		// Treat k as the structuring element's x coordinate
		for (int k = -maxVert; k <= maxVert; k++)
		{
			int maxHoriz = n / 2;
			// Treat l as the structuring element's y coordinate
			for (int l = -maxHoriz; l <= maxHoriz; l++)
			{
				// Make sure that the structuring element has a value of 1 in the position being processed,
				// and that the point with the structuring element overlaid is also inside the bounds of the image.
				if (d_structuringElement[(k + maxVert)*n + (l + maxHoriz)] == 1 && rowIndex + k >= 0 && rowIndex + k < h && colIndex + l >= 0 && colIndex + l < w)
				{
					// Determine offset [NOT USED HERE SINCE CUDA VERSION USES TEXTURE INSTEAD OF RAW MEMORY]
					//int offset = k*w*CHANNEL_COUNT + l*CHANNEL_COUNT;

					// Convert float4 texture info to uchar4 to extract r, g, b, and a components
					float4 texel = tex2D(rgbaTex, colIndex + l, rowIndex + k);
					// bytes.x = red, bytes.y = green, bytes.z = blue, bytes.w = alpha
					uchar4 bytes;
					convertTexFloatToUChar(&bytes, texel);
					//printf("r=%d, g=%d, b=%d, a=%d, texel=%d; ", bytes.x, bytes.y, bytes.z, bytes.w, texel);

					// Checks for dilation
					if (bytes.x > rMax)
						rMax = bytes.x;
					if (bytes.y > gMax)
						gMax = bytes.y;
					if (bytes.z > bMax)
						bMax = bytes.z;

					// Checks for erosion
					if (bytes.x < rMin)
						rMin = bytes.x;
					if (bytes.y < gMin)
						gMin = bytes.y;
					if (bytes.z < bMin)
						bMin = bytes.z;

					/*
					// The following method is for raw image memory manipulation by pointers and offsets
					// from the CPU version
					// Checks for dilation
					if ((*(id + index + offset)) > rMax)
						rMax = (unsigned char)(*(id + index + offset));
					if ((*(id + index + offset + 1)) > gMax)
						gMax = (unsigned char)(*(id + index + offset + 1));
					if ((*(id + index + offset + 2)) > bMax)
						bMax = (unsigned char)(*(id + index + offset + 2));

					// Checks for erosion
					if ((*(id + index + offset)) < rMin)
						rMin = (unsigned char)(*(id + index + offset));
					if ((*(id + index + offset + 1)) < gMin)
						gMin = (unsigned char)(*(id + index + offset + 1));
					if ((*(id + index + offset + 2)) < bMin)
						bMin = (unsigned char)(*(id + index + offset + 2));
					*/
				}
			}
		}
		// Filter according to which filter is selected
		switch(filter)
		{
			case(FilterTypeEnum::AlmostAReference):
				// This is very succinct and crisp and clear!  Mostly black, which outlines etched in sharp white
				// THE BEST OUT OF ALL OF THEM -- and, the closest to the reference Laplacian image provided
				*(od + index) = ((rMax + rMin) / 2 - bytesCenterPixel.x) >= 0 ? (unsigned char)((rMax + rMin) / 2 - bytesCenterPixel.x) : 0;
				(*(od + index + 1)) = ((gMax + gMin) / 2 - bytesCenterPixel.y) >= 0 ? (unsigned char)((gMax + gMin) / 2 - bytesCenterPixel.y) : 0;
				(*(od + index + 2)) = ((bMax + bMin) / 2 - bytesCenterPixel.z) >= 0 ? (unsigned char)((bMax + bMin) / 2 - bytesCenterPixel.z) : 0;
				break;

			case(FilterTypeEnum::AlmostFlattened):
				// Looks like very succinct three shades of gray
				// This is a luminosity-type conversion to grayscale
				unsigned char red = (unsigned char)((((rMax + rMin) / 2 - bytesCenterPixel.x)/2 + 255)*0.21);
				unsigned char green = (unsigned char)((((gMax + gMin) / 2 - bytesCenterPixel.y)/2 + 255)*0.72);
				unsigned char blue = (unsigned char)((((bMax + bMin) / 2 - bytesCenterPixel.z)/2 + 255)*0.07);
				//*dst = ((rMax + rMin) / 2 - *index) >= 0 ? red+green+blue : 0;
				//(*(dst + 1)) = ((gMax + gMin) / 2 - (*(index + 1))) >= 0 ? red+green+blue : 0;
				//(*(dst + 2)) = ((bMax + bMin) / 2 - (*(index + 2))) >= 0 ? red+green+blue : 0;
				*(od + index) = red + green + blue;
				(*(od + index + 1)) = red + green + blue;
				(*(od + index + 2)) = red + green + blue;
				break;

			case(FilterTypeEnum::AntiAliasingSmoothFuzz):
				// Excellent and very succinct outlines!  Colorizes to blue and yellow (BUT NOT IN THE
				// CUDA VERSION FOR SOME REASON!!)
				// This is the Laplacian according to http://www.mif.vu.lt/atpazinimas/dip/FIP/fip-Morpholo.html
				// which defines it as �(dilation+erosion-2*source).  
				// (Wow, the order of operations of the green and blue commands was mistaken in the CPU version,
				// which produced though a really cool filter effect--but oddly does not seem reproducible in
				// this CUDA version!)
				//(*(od + index + 1)) = (unsigned char)(gMax + gMin - 2 * bytesCenterPixel.y / 2);
				//(*(od + index + 2)) = (unsigned char)(bMax + bMin - 2 * bytesCenterPixel.z / 2);
				*(od + index) = (unsigned char)((rMax + rMin - 2*bytesCenterPixel.x)/2);
				(*(od + index + 1)) = (unsigned char)((gMax + gMin - 2* bytesCenterPixel.y)/2);
				(*(od + index + 2)) = (unsigned char)((bMax + bMin - 2* bytesCenterPixel.z)/2);
				break;

			case(FilterTypeEnum::FuzzInWideOutline):
				// This is wrong--used src instead of index, but it produces a unique result--
				// good gray outlines, though rest of image is fuzzy.  Src is the location
				// of the first pixel in the original CPU code, and its behavior is emulated
				// here by getting the texel at the 0,0 position.
				float4 texel2 = tex2D(rgbaTex, 0, 0);
				uchar4 bytes2;
				convertTexFloatToUChar(&bytes2, texel2);
				*(od + index) = ((rMax + rMin) / 2 - bytes2.x) >= 0 ? ((rMax + rMin) / 2 - bytes2.x) : 0;
				(*(od + index + 1)) = ((gMax + gMin) / 2 - bytes2.y) >= 0 ? ((gMax + gMin) / 2 - bytes2.y) : 0;
				(*(od + index + 2)) = ((bMax + bMin) / 2 - bytes2.z) >= 0 ? ((bMax + bMin) / 2 - bytes2.z) : 0;
				break;

			case(FilterTypeEnum::GhostEdges):
				// From imageJ (very similar to the clamping method below found in imageJ)
				*(od + index) = clamp(rMax - rMin + 128, 0, 255);
				(*(od + index + 1)) = clamp(gMax - gMin + 128, 0, 255);
				(*(od + index + 2)) = clamp(bMax - bMin + 128, 0, 255);
				break;

			case(FilterTypeEnum::InvisoWithWideOutlines):
				// Excellent results--mostly black except the outlines
				*(od + index) = ((rMax - rMin) / 2);
				(*(od + index + 1)) = ((gMax - gMin) / 2);
				(*(od + index + 2)) = ((bMax - bMin) / 2);
				break;

			case(FilterTypeEnum::MosaicInGray):
				// Now convert to grayscale using luminosity algorithm.
				// It produces kind of a grayscale mosaic.
				unsigned char red2 = (unsigned char)((rMax + rMin - 2 * bytesCenterPixel.x) / 2) * 0.21;
				// Interesting mistake!! (see order of operations of above compared with below)
				unsigned char green2 = (unsigned char)(gMax + gMin - 2 * bytesCenterPixel.y / 2) * 0.72;
				unsigned char blue2 = (unsigned char)(bMax + bMin - 2 * bytesCenterPixel.z / 2) * 0.07;
				unsigned char gray = red2 + green2 + blue2;
				*(od + index) = gray;
				(*(od + index + 1)) = gray;
				(*(od + index + 2)) = gray;
				break;

			case(FilterTypeEnum::PsychedelicLines):
				// Very similar to psychedelic lines, below
				*(od + index) = (unsigned char)((rMax + rMin) / 2 - bytesCenterPixel.x);
				(*(od + index + 1)) = (unsigned char)((gMax + gMin) / 2 - bytesCenterPixel.y);
				(*(od + index + 2)) = (unsigned char)((bMax + bMin) / 2 - bytesCenterPixel.z);
				break;

			case(FilterTypeEnum::PsychedelicMellowed):
				*(od + index) = ((rMax + rMin) / 2 - bytesCenterPixel.x) >= 0 ? ((rMax + rMin) / 2 - bytesCenterPixel.x) + 128 : 0;
				(*(od + index + 1)) = ((gMax + gMin) / 2 - bytesCenterPixel.y) >= 0 ? ((gMax + gMin) / 2 - bytesCenterPixel.y) + 128 : 0;
				(*(od + index + 2)) = ((bMax + bMin) / 2 - bytesCenterPixel.z) >= 0 ? ((bMax + bMin) / 2 - bytesCenterPixel.z) + 128 : 0;
				break;

			case(FilterTypeEnum::ReliefInGray):
				// Good results, and is very similar to the other SECOND BEST
				*(od + index) = clamp((((rMax + rMin) - 2* bytesCenterPixel.x)/2 + 255)/2, 0, 255);
				(*(od + index + 1)) = clamp((((gMax + gMin) -2* bytesCenterPixel.y)/2 + 255)/2, 0, 255);
				(*(od + index + 2)) = clamp((((bMax + bMin) -2* bytesCenterPixel.z)/2 + 255)/2, 0, 255);
				break;

			// The following filters produce good results, too, but in most cases are similar to the ones above

			// Wow, psychedelic lines!!!
			//*dst = clamp((rMax + rMin) / 2 - *index, 0, 255);
			//(*(dst + 1)) = clamp((gMax + gMin) / 2 - (*(index+1)), 0, 255);
			//(*(dst + 2)) = clamp((bMax + bMin) / 2 - (*(index+2)), 0, 255);

			// Almost a black and white result
			//*dst = (unsigned char)(((rMax + rMin) / 2 - *index) / 2 + 255);
			//(*(dst + 1)) = (unsigned char)(((gMax + gMin) / 2 - (*(index + 1))) / 2 + 255);
			//(*(dst + 2)) = (unsigned char)(((bMax + bMin) / 2 - (*(index + 2))) / 2 + 255);

			// This block will produce a negative of whatever filter is applied before it
			// Now try producing a negative of the Laplacian (or other--whichever is processed immediately
			// before this block), above (should be processed subsequently from it):
			//*dst = 255 - *dst;
			//(*(dst + 1)) = 255 - (*(dst + 1));
			//(*(dst + 2)) = 255 - (*(dst + 2));

			// This clamping mechanism was found in imageJ
			//unsigned char rExternalGradientDilation = clamp(rMax - *index, 0, 255);
			//unsigned char gExternalGradientDilation = clamp(gMax - *(index + 1), 0, 255);
			//unsigned char bExternalGradientDilation = clamp(bMax - *(index + 2), 0, 255);
			//unsigned char rInternalGradientErosion = clamp(rMin - *index, 0, 255);
			//unsigned char gInternalGradientErosion = clamp(gMin - *(index + 1), 0, 255);
			//unsigned char bInternalGradientErosion = clamp(bMin - *(index + 2), 0, 255);
			//*dst = (unsigned char)clamp(rExternalGradientDilation - rInternalGradientErosion + 128, 0, 255);
			//(*(dst + 1)) = (unsigned char)clamp(gExternalGradientDilation - gInternalGradientErosion + 128, 0, 255);
			//(*(dst + 2)) = (unsigned char)clamp(bExternalGradientDilation - bInternalGradientErosion + 128, 0, 255);

			//**** Wow, very good, all gray scale SECOND BEST
			//*dst = ((rMax + rMin) / 2 - *index) / 2 + 128;
			//(*(dst + 1)) = ((gMax + gMin) / 2 - (*(index + 1))) / 2 + 128;
			//(*(dst + 2)) = ((bMax + bMin) / 2 - (*(index + 2))) / 2 + 128;

			// Create luminescent bars
			//*(od + index) = (blockIdx.x*blockDim.x + threadIdx.x) % 256;
			//(*(od + index + 1)) = (blockIdx.x*blockDim.x + threadIdx.x) % 256;
			//(*(od + index + 2)) = (blockIdx.x*blockDim.x + threadIdx.x) % 256;
			//printf("r=%d, g=%d, b=%d; ", rMax, gMax, bMax);

			/*
			unsigned char red = (unsigned char)(((rMax + rMin) / 2 - *index)*0.21);
			unsigned char green = (unsigned char)(((gMax + gMin) / 2 - (*(index + 1)))*0.72);
			unsigned char blue = (unsigned char)(((bMax + bMin) / 2 - (*(index + 2)))*0.07);
			//*dst = ((rMax + rMin) / 2 - *index) >= 0 ? red+green+blue : 0;
			//(*(dst + 1)) = ((gMax + gMin) / 2 - (*(index + 1))) >= 0 ? red+green+blue : 0;
			//(*(dst + 2)) = ((bMax + bMin) / 2 - (*(index + 2))) >= 0 ? red+green+blue : 0;
			*dst = red + green + blue;
			(*(dst + 1)) = red + green + blue;
			(*(dst + 2)) = red + green + blue;
			*/
		}
	}
}

// RGBA version
extern "C"
double laplacianFilterRGBA(unsigned char *d_src, unsigned char *d_temp, unsigned char *d_dest, int width, int height,
	int iterations, int nthreads, StopWatchInterface *timer, StructuringElementEnum element, FilterTypeEnum filter) //int structuringElement[], int size)
{
	// Copy the array containing the structuring element into the device's memory
	// Gotta be an easier way to do this (would be nice if could use C++11 std::array, for example)
	// For some reason passing in the array from the host code doesn't work (see the backtracking
	// involved with the method signature, above)
	unsigned int n = 0;
	int *devArray;
	if (element == StructuringElementEnum::disk3x3)
	{
		n = int(sqrt(sizeof(disk3x3StructuringElement) / sizeof(*disk3x3StructuringElement)));
		checkCudaErrors(hipMalloc((void**)&devArray, n*n * sizeof(int)));
		checkCudaErrors(hipMemcpy(devArray, &disk3x3StructuringElement, n*n * sizeof(int), hipMemcpyHostToDevice));
	}
	if (element == StructuringElementEnum::disk5x5)
	{
		n = int(sqrt(sizeof(disk5x5StructuringElement) / sizeof(*disk5x5StructuringElement)));
		checkCudaErrors(hipMalloc((void**)&devArray, n*n * sizeof(int)));
		checkCudaErrors(hipMemcpy(devArray, &disk5x5StructuringElement, n*n * sizeof(int), hipMemcpyHostToDevice));
	}
	if (element == StructuringElementEnum::disk7x7)
	{
		n = int(sqrt(sizeof(disk7x7StructuringElement) / sizeof(*disk7x7StructuringElement)));
		checkCudaErrors(hipMalloc((void**)&devArray, n*n * sizeof(int)));
		checkCudaErrors(hipMemcpy(devArray, &disk7x7StructuringElement, n*n * sizeof(int), hipMemcpyHostToDevice));
	}
	else if (element == StructuringElementEnum::square3x3)
	{
		n = int(sqrt(sizeof(square3x3StructuringElement) / sizeof(*square3x3StructuringElement)));
		checkCudaErrors(hipMalloc((void**)&devArray, n*n * sizeof(int)));
		checkCudaErrors(hipMemcpy(devArray, &square3x3StructuringElement, n*n * sizeof(int), hipMemcpyHostToDevice));
	}
	else if (element == StructuringElementEnum::square5x5)
	{
		n = int(sqrt(sizeof(square5x5StructuringElement) / sizeof(*square5x5StructuringElement)));
		checkCudaErrors(hipMalloc((void**)&devArray, n*n * sizeof(int)));
		checkCudaErrors(hipMemcpy(devArray, &square5x5StructuringElement, n*n * sizeof(int), hipMemcpyHostToDevice));
	}
	else if (element == StructuringElementEnum::square7x7)
	{
		n = int(sqrt(sizeof(square7x7StructuringElement) / sizeof(*square7x7StructuringElement)));
		checkCudaErrors(hipMalloc((void**)&devArray, n*n * sizeof(int)));
		checkCudaErrors(hipMemcpy(devArray, &square7x7StructuringElement, n*n * sizeof(int), hipMemcpyHostToDevice));
	}
	else if (element == StructuringElementEnum::ring3x3)
	{
		n = int(sqrt(sizeof(ring3x3StructuringElement) / sizeof(*ring3x3StructuringElement)));
		checkCudaErrors(hipMalloc((void**)&devArray, n*n * sizeof(int)));
		checkCudaErrors(hipMemcpy(devArray, &ring3x3StructuringElement, n*n * sizeof(int), hipMemcpyHostToDevice));
	}
	else if (element == StructuringElementEnum::ring5x5)
	{
		n = int(sqrt(sizeof(ring5x5StructuringElement) / sizeof(*ring5x5StructuringElement)));
		checkCudaErrors(hipMalloc((void**)&devArray, n*n * sizeof(int)));
		checkCudaErrors(hipMemcpy(devArray, &ring5x5StructuringElement, n*n * sizeof(int), hipMemcpyHostToDevice));
	}
	else if (element == StructuringElementEnum::ring7x7)
	{
		n = int(sqrt(sizeof(ring7x7StructuringElement) / sizeof(*ring7x7StructuringElement)));
		checkCudaErrors(hipMalloc((void**)&devArray, n*n * sizeof(int)));
		checkCudaErrors(hipMemcpy(devArray, &ring7x7StructuringElement, n*n * sizeof(int), hipMemcpyHostToDevice));
	}

	checkCudaErrors(hipBindTextureToArray(rgbaTex, d_array));

	// var for kernel computation timing
	double dKernelTime;

	for (int i = 0; i<iterations; i++)
	{
		// sync host and start kernel computation timer_kernel
		dKernelTime = 0.0;
		checkCudaErrors(hipDeviceSynchronize());
		sdkResetTimer(&timer);

		// use texture for horizontal pass
		//d_boxfilter_rgba_x << < height / nthreads, nthreads, 0 >> >(d_temp, width, height, 10);
		//d_boxfilter_rgba_y << < width / nthreads, nthreads, 0 >> >(d_temp, d_dest, width, height, 10);

		dim3 dimBlock = dim3(16, 16);
		int yBlocks = width / dimBlock.y + ((width%dimBlock.y) == 0 ? 0 : 1);
		int xBlocks = height / dimBlock.x + ((height%dimBlock.x) == 0 ? 0 : 1);
		dim3 dimGrid = dim3(xBlocks, yBlocks);
		d_laplacianFilter_rgba <<< dimGrid, dimBlock >>>(d_temp, d_dest, width, height, filter, devArray, n);

		// sync host and stop computation timer_kernel
		checkCudaErrors(hipDeviceSynchronize());
		dKernelTime += sdkGetTimerValue(&timer);

		if (iterations > 1)
		{
			// copy result back from global memory to array
			checkCudaErrors(hipMemcpyToArray(d_tempArray, 0, 0, d_dest, width * height * sizeof(float), hipMemcpyDeviceToDevice));
			checkCudaErrors(hipBindTextureToArray(rgbaTex, d_tempArray));
		}
	}

	return ((dKernelTime / 1000.) / (double)iterations);
}

#endif // #ifndef _LAPLACIANFILTER_KERNEL_H_
